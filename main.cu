
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <iomanip>

#define _DEBUG

// from CUDA book
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//

template <typename T>
inline void printArray(const T *array, const int size) {
	for (int i = 0; i < size; ++i) {
		std::cout << array[i] << "\t";
	}
	std::cout << std::endl;
}

template <typename T>
inline void arrayMap(T *array, const int size, void (*callback)(T&) ) {
	for (int i = 0; i < size; ++i) {
		callback(array[i]);
	}
}

struct NodeData {
	float v_h;
	float G_0;
	float G_max;
	float v_d;
	float v_r;
	int   last_t;
	bool  send;
};

inline void printG(NodeData &data) {
	std::cout << std::fixed << std::setprecision(2) << data.G_0 << "\t";
}


__global__ void recv(const int N, const int *V, NodeData *Vdata, const int Elen, const int *E,
		int *M, const int t_c, const int t_p) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	NodeData *data = &Vdata[tid];

	if (data->send || tid >= N) {
		return;
	}

	int lastTime = 0, msgCount = 0;
	int start = V[tid];
	int end = V[tid + 1], msg;

	// reading messages
	for (int i = start; i < end; ++i) {
		msg = M[i];

		if (msg <= 0 || msg <= data->last_t) {
			continue;
		}

		if (lastTime < msg) {
			lastTime = msg;
		}
		++msgCount;
	}

	data->last_t = lastTime;

	// processing messages
	data->G_0 = data->G_max - (data->G_max - data->G_0) * exp(-0.01 * msgCount * t_p);

}

__global__ void send(const int N, const int *V, NodeData *Vdata, const int Elen, const int *E,
		int *M, const int t_c, const int t_p) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	NodeData *data = &Vdata[tid];

	if (data->send || tid >= N) {
		return;
	}

	int start = V[tid];
	int end = V[tid + 1];
	int lastTime, start2, end2, v;

	// sending messages
	if (data->G_0 * (data->v_h + data->v_r) >= data->v_d) {
		data->send = true;
		lastTime = data->last_t + t_p + t_c;

		for (int i = start; i < end; ++i) {
			v = E[i];
			start2 = V[v];
			end2 = V[v + 1];
			for (int j = start2; j < end2; ++j) {
				if (E[j] == tid) {
					M[j] = lastTime;
					break;
				}
			}
		}
	}
}


int main(void) {

	// read parameters
	int N,				// no. of vertices (nodes)
		Elen,			// no. of edges
		Vsrc;			// source vertex
	int t_c = 3,		// communication time [s]
		t_p = 30,		// processing time [s] e.g. short movie
		t_s = 330;		// max. simulation time [s]
	float v_h = 0,		// reflection potential [-1; 1]
	    G_0 = 1,		// initial conductivity [0, G_max)
	    G_max = 100,	// max. conductivity (G_0, +inf)
	    v_r = 1,		// registration potential (0, +inf)
	    v_d = 1;		// decision-making potential (0, +inf)

	int *V, *dev_V,
		*E, *dev_E,
		*M, *dev_M;
	NodeData *Vdata, *dev_Vdata;

#ifdef _DEBUG
	std::cout << "Program started." << std::endl;
#endif

	//
	std::cin >> N;

	// N+1 - space for one extra element at the end (for easiest iteration through graph)
	V = (int*) malloc((N+1) * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_V, (N+1) * sizeof(int)));

	Vdata = (NodeData*) malloc(N * sizeof(NodeData));
	HANDLE_ERROR(hipMalloc((void**) &dev_Vdata, N * sizeof(NodeData)));

	for (int i = 0; i < N; ++i) {
		std::cin >> v_h >> G_0 >> G_max >> v_d;
		Vdata[i].v_h = v_h;
		Vdata[i].G_0 = G_0;
		Vdata[i].G_max = G_max;
		Vdata[i].v_d = v_d;
		Vdata[i].v_r = v_r;
		Vdata[i].last_t = 0;
		Vdata[i].send = false;
	}

	//
	std::cin >> Elen;

	E = (int*) malloc(Elen * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_E, Elen * sizeof(int)));

	M = (int*) calloc(Elen, sizeof(int));	// zero-initialized
	HANDLE_ERROR(hipMalloc((void**) &dev_M, Elen * sizeof(int)));

	V[0] = 0;
	{
		int Elen_i, start, end, e_i;
		for (int i = 0; i < N; ++i) {
			std::cin >> Elen_i;
			start = V[i];
			end = start + Elen_i;
			V[i + 1] = end;

			for (int j = start; j < end; ++j) {
				std::cin >> e_i;
				E[j] = e_i;
			}
		}
	}

	// get source vector and create M(0)
	std::cin >> Vsrc;
	Vdata[Vsrc].send = true;

	{
		int start = V[Vsrc];
		int end = V[Vsrc + 1];
		int start2, end2, v;

		for (int i = start; i < end; ++i) {
			v = E[i];
			start2 = V[v];
			end2 = V[v + 1];
			for (int j = start2; j < end2; ++j) {
				if (E[j] == Vsrc) {
					M[j] = 1;
					break;
				}
			}
		}
	}

	//
	std::cin >> t_c >> t_p >> t_s;

#ifdef _DEBUG
	std::cout << "V:\t";
	printArray(V, N);

	std::cout << "E:\t";
	printArray(E, Elen);

	std::cout << "M_0:\t";
	printArray(M, Elen);
#endif

	//
	const int threadsPerBlock = 1024;
	int blocksPerGrid = (N + threadsPerBlock-1) / threadsPerBlock;	// floor

	// copy the data to the GPU
	HANDLE_ERROR(hipMemcpy(dev_V, V, (N+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_E, E, Elen * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_M, M, Elen * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_Vdata, Vdata, N * sizeof(NodeData), hipMemcpyHostToDevice));

	for (int t = 1; t <= t_s; t += t_c + t_p) {
		recv<<<blocksPerGrid, threadsPerBlock>>>(N, dev_V, dev_Vdata, Elen, dev_E, dev_M, t_c, t_p);
		send<<<blocksPerGrid, threadsPerBlock>>>(N, dev_V, dev_Vdata, Elen, dev_E, dev_M, t_c, t_p);

#ifdef _DEBUG
		HANDLE_ERROR(hipMemcpy(M, dev_M, Elen * sizeof(int), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(Vdata, dev_Vdata, N * sizeof(NodeData), hipMemcpyDeviceToHost));

		std::cout << "M_" << (t / (t_c + t_p) + 1) << ":\t";
		printArray(M, Elen);

		std::cout << "G_" << (t / (t_c + t_p) + 1) << ":\t";
		arrayMap(Vdata, N, printG);
		std::cout << std::endl;
#endif
	}

	// copy the data from the GPU to the CPU
	HANDLE_ERROR(hipMemcpy(M, dev_M, Elen * sizeof(int), hipMemcpyDeviceToHost));

//#ifdef _DEBUG
//	std::cout << "M: ";
//	printArray(M, Elen);
//#endif

	// generate output
	// 1. how many recipients
	// 2. max. distance from source (range)
	// 3. a graph at the end of the simulation in dot format
	//@todo: ^^

	// free memory on the GPU side
	hipFree(dev_V);
	hipFree(dev_E);
	hipFree(dev_M);
	hipFree(dev_Vdata);

	// free memory on the CPU side
	free(V);
	free(E);
	free(M);
	free(Vdata);
}
