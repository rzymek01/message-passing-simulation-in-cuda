/**
 * Usage:
 * ./msg-spr-sim [<no_of_threads_per_block>] [<device_id>] <”<input_file>” >”<output_file>” 2>”<file_with_exec_time>”
 * ./msg-spr-sim [...] | dot -Tpng -o graph.png
 *
 * Input file format:
 * 		| %d			// #V
 * x #V	| %f %f %f %f 	// v_h_i, G_0_i, G_max_i, v_d_i
 * 		| %d			// #E
 * x #V	| %d [%d, ...]	// #edges of v_i [v_j, ...]
 * 		| %d			// source v_i
 * 		| %d %d %d 		// t_c, t_p, t_s
 *
 * Output file format: dot
 * Stderr output: single double number - execution time
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <iomanip>

//#define _DEBUG

// from CUDA book
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << "in " << file << " at " << line << std::endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//

template <typename T>
inline void printArray(const T *array, const int size) {
	for (int i = 0; i < size; ++i) {
		std::cout << array[i] << "\t";
	}
	std::cout << std::endl;
}

template <typename T>
inline void arrayMap(T *array, const int size, void (*callback)(T&) ) {
	for (int i = 0; i < size; ++i) {
		callback(array[i]);
	}
}

struct NodeData {
	float v_h;
	float G_0;
	float G_max;
	float v_d;
	float v_r;
	int   last_t;
	int   new_last_t;
	bool  send;
	bool  new_send;
};

inline void printG(NodeData &data) {
	std::cout << std::fixed << std::setprecision(2) << data.G_0 << "\t";
}

__global__ void recv(const int N, const int *V, NodeData *Vdata, const int Elen,
		const int *X, const int *M, const int t_p) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= Elen) {
		return;
	}

	int msg = M[tid];
	NodeData *data = &Vdata[X[tid]];

	data->send = data->new_send;

	if (msg <= 0 || msg <= data->last_t) {
		return;
	}

	// determine new last time
	atomicMax(&(data->new_last_t), msg);

	// change G_0
	float G_0 = data->G_0, G_max = data->G_max;
	atomicExch(&(data->G_0), G_max - (G_max - G_0) * exp(-0.01 * t_p));
}

__global__ void send(const int N, const int *V, NodeData *Vdata, const int Elen,
		const int *E, int *M, const int t_c, const int t_p) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= Elen) {
		return;
	}

	NodeData *data = &Vdata[E[tid]];

	if (data->send) {
		return;
	}

	data->last_t = data->new_last_t;

	// sending messages
	if (data->G_0 * (data->v_h + data->v_r) >= data->v_d) {
		data->new_send = true;
		M[tid] = data->new_last_t + t_p + t_c;
	}
}


int main(int argc, char* argv[]) {

	// obtain available devices
	int deviceCount;
	HANDLE_ERROR(hipGetDeviceCount(&deviceCount));

	if (deviceCount <= 0) {
		std::cerr << "No CUDA devices has been found!" << std::endl;
		exit(1);
	}

	hipDeviceProp_t *prop = new hipDeviceProp_t[deviceCount];

	for (int i = 0; i < deviceCount; ++i) {
		HANDLE_ERROR(hipGetDeviceProperties(&(prop[i]), i));
#ifdef _DEBUG
		std::cout << "/*name: " << prop[i].name << "; max_threads_per_block: " << prop[i].maxThreadsPerBlock << "*/\n";
#endif
	}

	// read parameters
	int threadsPerBlock = -1,
		deviceId = -1;

	if (argc >= 2) {
		threadsPerBlock = atoi(argv[1]);
	}
	if (argc >= 3) {
		deviceId = atoi(argv[2]);
	}

	if (deviceId == -1) {
		deviceId = 0;
	}
	if (threadsPerBlock == -1) {
		threadsPerBlock = prop[deviceId].maxThreadsPerBlock;
	}

	if (deviceId < 0 || deviceId >= deviceCount) {
		std::cerr << "Device id out of range!" << std::endl;
		exit(3);
	}
	if (threadsPerBlock < 2 || threadsPerBlock > prop[deviceId].maxThreadsPerBlock) {
		std::cerr << "Number of threads per block is too small or too big!" << std::endl;
		exit(2);
	}

	// select device
	hipSetDevice(deviceId);

	std::cout << "/*selected device: " << prop[deviceId].name << "; threads_per_block: "
			<< threadsPerBlock << "*/\n";

	// read input data
	int N,				// no. of vertices (nodes)
		Elen,			// no. of edges
		Vsrc;			// source vertex
	int t_c = 3,		// communication time [s]
		t_p = 30,		// processing time [s] e.g. short movie
		t_s = 330;		// max. simulation time [s]
	float v_h = 0,		// reflection potential [-1; 1]
	    G_0 = 1,		// initial conductivity [0, G_max)
	    G_max = 100,	// max. conductivity (G_0, +inf)
	    v_r = 1,		// registration potential (0, +inf)
	    v_d = 1;		// decision-making potential (0, +inf)

	int *V, *dev_V,
		*E, *dev_E,
		*M, *dev_M, *dev_X;
	NodeData *Vdata, *dev_Vdata;

#ifdef _DEBUG
	std::cout << "/*\nProgram started." << std::endl;
#endif

	//
	std::cin >> N;

	// N+1 - space for one extra element at the end (for easiest iteration through graph)
	V = (int*) malloc((N+1) * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_V, (N+1) * sizeof(int)));

	Vdata = (NodeData*) malloc(N * sizeof(NodeData));
	HANDLE_ERROR(hipMalloc((void**) &dev_Vdata, N * sizeof(NodeData)));

	for (int i = 0; i < N; ++i) {
		std::cin >> v_h >> G_0 >> G_max >> v_d;
		Vdata[i].v_h = v_h;
		Vdata[i].G_0 = G_0;
		Vdata[i].G_max = G_max;
		Vdata[i].v_d = v_d;
		Vdata[i].v_r = v_r;
		Vdata[i].last_t = 0;
		Vdata[i].new_last_t = 0;
		Vdata[i].send = false;
		Vdata[i].new_send = false;
	}

	//
	std::cin >> Elen;

	E = (int*) malloc(Elen * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_E, Elen * sizeof(int)));

	M = (int*) malloc(Elen * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_M, Elen * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_X, Elen * sizeof(int)));

	V[0] = 0;
	{
		int Elen_i, start, end, e_i;
		for (int i = 0; i < N; ++i) {
			std::cin >> Elen_i;
			start = V[i];
			end = start + Elen_i;
			V[i + 1] = end;

			for (int j = start; j < end; ++j) {
				std::cin >> e_i;
				E[j] = e_i;
				M[j] = i; // M is used here as X (X is used only in kernel)
			}
		}
	}

//	printArray(M, Elen);

	HANDLE_ERROR(hipMemcpy(dev_X, M, Elen * sizeof(int), hipMemcpyHostToDevice));

	// zero-filling M
	memset((void*)M, 0, Elen * sizeof(int));

//	printArray(M, Elen);

	// get source vector and create M(0)
	std::cin >> Vsrc;
	Vdata[Vsrc].send = true;
	Vdata[Vsrc].new_send = true;

	{
		int start = V[Vsrc];
		int end = V[Vsrc + 1];
		int start2, end2, v;

		for (int i = start; i < end; ++i) {
			v = E[i];
			start2 = V[v];
			end2 = V[v + 1];
			for (int j = start2; j < end2; ++j) {
				if (E[j] == Vsrc) {
					M[j] = 1;
					break;
				}
			}
		}
	}

	//
	std::cin >> t_c >> t_p >> t_s;

#ifdef _DEBUG
	std::cout << "V:\t";
	printArray(V, N);

	std::cout << "E:\t";
	printArray(E, Elen);

	std::cout << "M_0:\t";
	printArray(M, Elen);
#endif

	//
	const int blocksPerGrid = (Elen + threadsPerBlock-1) / threadsPerBlock;	// ceil(x/y)

	// capture the start time
	hipEvent_t	startEvent, stopEvent;
	HANDLE_ERROR(hipEventCreate(&startEvent));
	HANDLE_ERROR(hipEventCreate(&stopEvent));
	HANDLE_ERROR(hipEventRecord(startEvent, 0));

	// copy the data to the GPU
	HANDLE_ERROR(hipMemcpy(dev_V, V, (N+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_E, E, Elen * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_M, M, Elen * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_Vdata, Vdata, N * sizeof(NodeData), hipMemcpyHostToDevice));

	for (int t = 1, i = 1; t <= t_s; t += t_c + t_p, ++i) {
		recv<<<blocksPerGrid, threadsPerBlock>>>(N, dev_V, dev_Vdata, Elen, dev_X, dev_M, t_p);
		send<<<blocksPerGrid, threadsPerBlock>>>(N, dev_V, dev_Vdata, Elen, dev_E, dev_M, t_c, t_p);
	}

	// copy the data from the GPU to the CPU
	HANDLE_ERROR(hipMemcpy(M, dev_M, Elen * sizeof(int), hipMemcpyDeviceToHost));

	// capture the end time
	HANDLE_ERROR(hipEventRecord(stopEvent, 0));
	hipEventSynchronize(stopEvent);

	float compTime;
	HANDLE_ERROR(hipEventElapsedTime(&compTime, startEvent, stopEvent));

	HANDLE_ERROR(hipEventDestroy(startEvent));
	HANDLE_ERROR(hipEventDestroy(stopEvent));

#ifdef _DEBUG
	std::cout << "M: ";
	printArray(M, Elen);
	std::cout << "*/\n";
#endif

	// write computing time (to cerr for simplicity)

	std::cerr << compTime << std::endl;

	// generate output
	// 1. how many recipients
	// 2. max. distance from source (range)
	// -> 3. a graph at the end of the simulation in dot format

	std::cout << "digraph G {\n";
	std::cout << "\tnode [fontsize=12]\n";
	std::cout << "\tedge [fontcolor=\"0.5 0.5 0.5\",fontsize=8]\n";
	std::cout << "\t" << Vsrc << " [label=\"src\"]\n\n";

	int start, end;
	for (int i = 0; i < N; ++i) {
		start = V[i];
		end = V[i + 1];
		for (int j = start; j < end; ++j) {
			std::cout << "\t" << E[j] << " -> " << i;
			if (M[j] > 0) {
				std::cout << " [label=" << M[j] << "]\n";
			} else {
				std::cout << " [style=dotted]\n";
			}
		}
	}


	std::cout << "}" << std::endl;

	// free memory on the GPU side
	hipFree(dev_V);
	hipFree(dev_E);
	hipFree(dev_M);
	hipFree(dev_Vdata);

	// free memory on the CPU side
	free(V);
	free(E);
	free(M);
	free(Vdata);
}
